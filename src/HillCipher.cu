#include "hip/hip_runtime.h"
#include "HillCipher.h"

__global__ void matrixMultiplication( unsigned char *newVect, int *key, unsigned char *resultado, int keySize ){

    extern __shared__ int cache[];//Determina que es memoria que solo conocen los hilos del mismo bloque y se genera la variable cache por cada conjunto(Lo realiza el primer hilo en llegar)

    //Index para el elemento del vector
    int vectId = 0;

    //Index para la posicion del elemento de la llave
    int keyId = 0;

    int mod = 0;

    //Indexacion para la llave bidimensional
    keyId = ((threadIdx.x * keySize)+ threadIdx.y);

    //Indexacion para el vectorAudio
    vectId = ((blockIdx.x * keySize) + threadIdx.y);

    //Se realiza la multiplicacion de matrices renglon * columna
    mod = (key[(keyId)] * newVect[(vectId)]);

    //Se realiza una variable cache por cada bloque de hilos
    cache[(threadIdx.x * keySize) + threadIdx.y] = mod;

    __syncthreads();//Se sicronizan los hilos
    //Una vez que todos los hilos terminan se puede realizar la suma



    //Se va recorriendo el cache
    if (threadIdx.y  == 0)  // soy el hilo 0
    {
        int suma = 0;
        int j = 0;

        for ( j = 0; j < keySize; j++){
            //Realizamos la sumatoria de la cantidad de numeros correspondidos por el tamLlave(numColumnas) para
            //despues asignar el valor a la posicion que le corresponde al arregle resultado
            suma += cache[(threadIdx.x * keySize) + j];
        }

    //El resultado se almacenara en la posicion que indique su propia indexaxion, antes de almacenarlo se aplicara un mod 256
    resultado[blockIdx.x*blockDim.x+threadIdx.x] = (suma % 256);

    }
}

size_t HillCipher::next_chunk(){
    return fread( chunk, sizeof(unsigned char), buffer_size, file);
}

void HillCipher::clear_chunks(){
    hipMemset( chunk, 0, buffer_size);
    hipMemset( newChunk, 0, buffer_size);
}

void HillCipher::copy_key( int *key_vector ){
    /*for( int i=0; i<key_size*key_size; ++i ){
        key[i]=key_vector[i];
    }*/
    hipMemcpy(key, key_vector, sizeof(int) * key_size * key_size, hipMemcpyHostToDevice );
}

unsigned char HillCipher::create_output( std::string output_str ){
    //output=NULL;
    output=fopen(output_str.c_str(), "wb" );
    if( output == NULL ){
        printf("Couldn't create output file %s.\n", output_str.c_str());
        return 1;
    }
    return 0;
}

int HillCipher::next_multiple(int bytes){
    if ( bytes % key_size == 0 )
        return bytes;
    else
        return ( key_size - ( bytes % key_size ) ) + bytes;
}

void HillCipher::apply_key( int *key_vector, Tochfile &tochkey  ){
    //CUDA CONFIG
    dim3 dimBlock( key_size, key_size );
    copy_key(key_vector);
    hipDeviceSynchronize();
    size_t bytes_readed; 
    while( (bytes_readed=next_chunk()) > 0 ){
        if( bytes_readed != buffer_size ){
            hipMemset( chunk+bytes_readed, 0, buffer_size-bytes_readed);
        }
        bytes_readed=next_multiple(bytes_readed);
        dim3 dimGrid(bytes_readed);

        //Ejecutar el kernel paralelo
        matrixMultiplication << <dimGrid, dimBlock, sizeof(int)*(key_size*key_size)>> >( chunk, key, newChunk, key_size );
        hipDeviceSynchronize();
        if( bytes_readed != buffer_size ){
            hipMemset( newChunk+bytes_readed, 0, buffer_size-bytes_readed);
        }

        //Write back to file
        printf("Writing to file...\n");
        if( bytes_readed != buffer_size ) printf("Last byte: %d %d     bytes read:%zu\n", chunk[bytes_readed-1], newChunk[bytes_readed-1], bytes_readed);
        tochkey.write(newChunk, bytes_readed);
    }
    tochkey.close();
    fclose(output);
    clear_chunks();
    output=NULL;

}
void HillCipher::set_key_size(int size){
    key_size=size;
}


void HillCipher::read_file( char *path_to_file ){
    //Try to open the file
    FILE* file_ptr = fopen( path_to_file, "rb" );
    if (file_ptr == NULL)
    {
        fprintf( stderr, "Unable to open file: %s\n", path_to_file );
        return;
    }
    printf("Reading %s\n",path_to_file);
    file = file_ptr;
    file_size = getFileSize(file);
    filename = std::string(basename(path_to_file));
}

HillCipher::HillCipher(  ){
    //Set defaults
    set_defaults();

    //Reserve memory for the chunk 
    hipMallocManaged( &chunk, sizeof(unsigned char) * buffer_size );
    hipMallocManaged( &newChunk, sizeof(unsigned char) * buffer_size );
    hipMallocManaged( &key, sizeof(int) * key_size * key_size );
}

HillCipher::HillCipher( char *path_to_file ){
    //Set defaults
    set_defaults();

    //Reserve memory for the chunk
    hipMallocManaged( &chunk, sizeof(unsigned char) * buffer_size );
    hipMallocManaged( &newChunk, sizeof(unsigned char) * buffer_size );
    hipMallocManaged( &key, sizeof(int) * key_size * key_size );
    //Read file
    read_file(path_to_file);
}



HillCipher::~HillCipher(){
    if( file != NULL )
        fclose(file);
    file=NULL;

    if( chunk != NULL )
        hipFree(chunk);
    if( key != NULL )
        hipFree(key);
    if( newChunk != NULL )
        hipFree(newChunk);
        
}

void HillCipher::print(){
    if( file_size > 0 )
        printf("Filename: %s\nFile size: %zu\n", filename.c_str(), file_size );
}


// find the file size
size_t HillCipher::getFileSize(FILE* inFile)
{
    size_t fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}

void HillCipher::set_defaults(){
    //Set attributes to default
    file_size=0;
    buffer_size = 1024 * 1024 * 32;
    //buffer_size = 32;
    file=NULL;
    chunk=NULL;
    mod=MOD;
    key_size=8;

}

/*Super formula nextInt = (cM - (tamAudio % cM)) + tamAudio;
<<El comun multiplo esta dado por cM = (tamAudio * tamLlave)>> */
size_t nextInt(int vect_size, int key_size)
{
    int cm=vect_size * key_size;
    return (cm - (vect_size % cm)) + vect_size;
}
   ;